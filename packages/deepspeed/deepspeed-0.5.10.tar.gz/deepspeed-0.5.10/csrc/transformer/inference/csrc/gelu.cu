#include "hip/hip_runtime.h"
#include "custom_cuda_layers.h"

inline __device__ float gelu(const float x)
{
    const float sqrt_param = 0.79788456080286535587989211986876f;
    const float mul_param = 0.044715;
    return x * 0.5f * (1.0f + tanhf(sqrt_param * (x + mul_param * x * x * x)));
}

__global__ void fused_bias_gelu(float* input,
                                const float* bias,
                                int total_count,
                                int intermediate_size)
{
    float4* input_cast = reinterpret_cast<float4*>(input);
    const float4* bias_cast = reinterpret_cast<const float4*>(bias);
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float4 data = input_cast[offset];
        float4 bias_data = bias_cast[offset % intermediate_size];

        data.x += bias_data.x;
        data.y += bias_data.y;
        data.z += bias_data.z;
        data.w += bias_data.w;

        data.x = gelu(data.x);
        data.y = gelu(data.y);
        data.z = gelu(data.z);
        data.w = gelu(data.w);

        input_cast[offset] = data;
    }
}

__global__ void fused_bias_gelu(__half* input,
                                const __half* bias,
                                int total_count,
                                int intermediate_size)
{
#if __CUDA_ARCH__ >= 700

    float2* input_cast = reinterpret_cast<float2*>(input);
    const float2* bias_cast = reinterpret_cast<const float2*>(bias);

    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float2 vals_vec = input_cast[offset];
        float2 bias_vec = bias_cast[offset % intermediate_size];

        __half2* vals_half = reinterpret_cast<__half2*>(&vals_vec);
        __half2* bias_half = reinterpret_cast<__half2*>(&bias_vec);

        float2 low_data = __half22float2(vals_half[0]);
        float2 high_data = __half22float2(vals_half[1]);

        float2 low_bias = __half22float2(bias_half[0]);
        float2 high_bias = __half22float2(bias_half[1]);

        low_data.x += low_bias.x;
        low_data.y += low_bias.y;
        high_data.x += high_bias.x;
        high_data.y += high_bias.y;

        low_data.x = gelu(low_data.x);
        low_data.y = gelu(low_data.y);
        high_data.x = gelu(high_data.x);
        high_data.y = gelu(high_data.y);

        vals_half[0] = __float22half2_rn(low_data);
        vals_half[1] = __float22half2_rn(high_data);

        input_cast[offset] = vals_vec;
    }
#endif
}

template <typename T>
void launch_bias_gelu(T* input,
                      const T* bias,
                      int intermediate_size,
                      int batch_size,
                      hipStream_t stream)
{
    int total_count = batch_size * (intermediate_size / 4);
    int threads = 1024;  // intermediate_size / iterations / 4;
    dim3 block_dims(threads);
    dim3 grid_dims(((total_count - 1) / 1024 + 1));  // (batch_size);

    fused_bias_gelu<<<grid_dims, block_dims, 0, stream>>>(
        input, bias, total_count, intermediate_size / 4);
}

template void launch_bias_gelu<float>(float*, const float*, int, int, hipStream_t);
template void launch_bias_gelu<__half>(__half*, const __half*, int, int, hipStream_t);

__global__ void fused_bias_add(float* input, const float* bias, int total_count, int hidden_size)
{
    float4* input_cast = reinterpret_cast<float4*>(input);
    const float4* bias_cast = reinterpret_cast<const float4*>(bias);
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float4 data = input_cast[offset];
        float4 bias_data = bias_cast[offset % hidden_size];

        data.x += bias_data.x;
        data.y += bias_data.y;
        data.z += bias_data.z;
        data.w += bias_data.w;

        input_cast[offset] = data;
    }
}

__global__ void fused_bias_add(__half* input, const __half* bias, int total_count, int hidden_size)
{
#if __CUDA_ARCH__ >= 700

    float2* input_cast = reinterpret_cast<float2*>(input);
    const float2* bias_cast = reinterpret_cast<const float2*>(bias);

    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float2 vals_vec = input_cast[offset];
        float2 bias_vec = bias_cast[offset % hidden_size];

        __half2* vals_half = reinterpret_cast<__half2*>(&vals_vec);
        __half2* bias_half = reinterpret_cast<__half2*>(&bias_vec);

        float2 low_data = __half22float2(vals_half[0]);
        float2 high_data = __half22float2(vals_half[1]);

        float2 low_bias = __half22float2(bias_half[0]);
        float2 high_bias = __half22float2(bias_half[1]);

        low_data.x += low_bias.x;
        low_data.y += low_bias.y;
        high_data.x += high_bias.x;
        high_data.y += high_bias.y;

        vals_half[0] = __float22half2_rn(low_data);
        vals_half[1] = __float22half2_rn(high_data);

        input_cast[offset] = vals_vec;
    }
#endif
}

template <typename T>
void launch_bias_add(T* input, const T* bias, int hidden_size, int batch_size, hipStream_t stream)
{
    int total_count = batch_size * (hidden_size / 4);
    int threads = 1024;  // hidden_size / iterations / 4;
    dim3 block_dims(threads);
    dim3 grid_dims(((total_count - 1) / threads + 1));  // (batch_size);

    fused_bias_add<<<grid_dims, block_dims, 0, stream>>>(input, bias, total_count, hidden_size / 4);
}

template void launch_bias_add<float>(float*, const float*, int, int, hipStream_t);
template void launch_bias_add<__half>(__half*, const __half*, int, int, hipStream_t);

__global__ void fused_bias_residual(float* input,
                                    const float* residual,
                                    const float* bias,
                                    int total_count,
                                    int intermediate_size)
{
    float4* input_cast = reinterpret_cast<float4*>(input);
    const float4* residual_cast = reinterpret_cast<const float4*>(residual);
    const float4* bias_cast = reinterpret_cast<const float4*>(bias);
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float4 data = input_cast[offset];
        float4 res_vec = residual_cast[offset];
        float4 bias_data = bias_cast[offset % intermediate_size];

        data.x += (res_vec.x + bias_data.x);
        data.y += (res_vec.y + bias_data.y);
        data.z += (res_vec.z + bias_data.z);
        data.w += (res_vec.w + bias_data.w);

        input_cast[offset] = data;
    }
}

__global__ void fused_bias_residual(__half* input,
                                    const __half* residual,
                                    const __half* bias,
                                    int total_count,
                                    int intermediate_size)
{
#if __CUDA_ARCH__ >= 700

    float2* input_cast = reinterpret_cast<float2*>(input);
    const float2* residual_cast = reinterpret_cast<const float2*>(residual);

    const float2* bias_cast = reinterpret_cast<const float2*>(bias);

    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float2 vals_vec = input_cast[offset];
        float2 res_vec = residual_cast[offset];

        float2 bias_vec = bias_cast[offset % intermediate_size];

        __half2* vals_half = reinterpret_cast<__half2*>(&vals_vec);
        __half2* res_half = reinterpret_cast<__half2*>(&res_vec);
        __half2* bias_half = reinterpret_cast<__half2*>(&bias_vec);

        float2 low_data = __half22float2(vals_half[0]);
        float2 high_data = __half22float2(vals_half[1]);

        float2 low_res = __half22float2(res_half[0]);
        float2 high_res = __half22float2(res_half[1]);

        float2 low_bias = __half22float2(bias_half[0]);
        float2 high_bias = __half22float2(bias_half[1]);

        low_data.x += (low_res.x + low_bias.x);
        low_data.y += (low_res.y + low_bias.y);
        high_data.x += (high_res.x + high_bias.x);
        high_data.y += (high_res.y + high_bias.y);

        vals_half[0] = __float22half2_rn(low_data);
        vals_half[1] = __float22half2_rn(high_data);

        input_cast[offset] = vals_vec;
    }
#endif
}

template <typename T>
void launch_bias_residual(T* input,
                          const T* residual,
                          const T* bias,
                          int batch,
                          int intermediate_size,
                          hipStream_t stream)
{
    int total_count = batch * intermediate_size / 4;
    dim3 block_dims(1024);
    dim3 grid_dims((total_count - 1) / 1024 + 1);  // (batch_size);

    fused_bias_residual<<<grid_dims, block_dims, 0, stream>>>(
        input, residual, bias, total_count, intermediate_size / 4);
}

template void launch_bias_residual<float>(float*,
                                          const float*,
                                          const float*,
                                          int,
                                          int,
                                          hipStream_t);
template void launch_bias_residual<__half>(__half*,
                                           const __half*,
                                           const __half*,
                                           int,
                                           int,
                                           hipStream_t);

__global__ void gptj_residual_add(float* input,
                                  float* output,
                                  float* attn,
                                  float* bias,
                                  int total_count,
                                  int intermediate_size)
{
    float4* input_cast = reinterpret_cast<float4*>(input);
    float4* output_cast = reinterpret_cast<float4*>(output);
    float4* attn_cast = reinterpret_cast<float4*>(attn);
    float4* bias_cast = reinterpret_cast<float4*>(bias);
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float4 data = input_cast[offset];
        float4 out = output_cast[offset];
        float4 res_vec = attn_cast[offset];
        float4 bias_data = bias_cast[offset % intermediate_size];

        data.x += (out.x + res_vec.x + bias_data.x);
        data.y += (out.y + res_vec.y + bias_data.y);
        data.z += (out.z + res_vec.z + bias_data.z);
        data.w += (out.w + res_vec.w + bias_data.w);

        output_cast[offset] = data;
    }
}

__global__ void gptj_residual_add(__half* input,
                                  __half* output,
                                  __half* attn,
                                  __half* bias,
                                  int total_count,
                                  int intermediate_size)
{
#if __CUDA_ARCH__ >= 700

    float2* input_cast = reinterpret_cast<float2*>(input);
    float2* output_cast = reinterpret_cast<float2*>(output);
    float2* attn_cast = reinterpret_cast<float2*>(attn);

    float2* bias_cast = reinterpret_cast<float2*>(bias);

    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float2 vals_vec = input_cast[offset];
        float2 out_vec = output_cast[offset];
        float2 res_vec = attn_cast[offset];

        float2 bias_vec = bias_cast[offset % intermediate_size];

        __half2* vals_half = reinterpret_cast<__half2*>(&vals_vec);
        __half2* out_half = reinterpret_cast<__half2*>(&out_vec);
        __half2* res_half = reinterpret_cast<__half2*>(&res_vec);
        __half2* bias_half = reinterpret_cast<__half2*>(&bias_vec);

        float2 low_data = __half22float2(vals_half[0]);
        float2 high_data = __half22float2(vals_half[1]);

        float2 low_out = __half22float2(out_half[0]);
        float2 high_out = __half22float2(out_half[1]);

        float2 low_res = __half22float2(res_half[0]);
        float2 high_res = __half22float2(res_half[1]);

        float2 low_bias = __half22float2(bias_half[0]);
        float2 high_bias = __half22float2(bias_half[1]);

        low_data.x += (low_out.x + low_res.x + low_bias.x);
        low_data.y += (low_out.y + low_res.y + low_bias.y);
        high_data.x += (high_out.x + high_res.x + high_bias.x);
        high_data.y += (high_out.y + high_res.y + high_bias.y);

        vals_half[0] = __float22half2_rn(low_data);
        vals_half[1] = __float22half2_rn(high_data);

        output_cast[offset] = vals_vec;
    }
#endif
}

template <typename T>
void launch_gptj_residual_add(T* input,
                              T* output,
                              T* attn,
                              T* bias,
                              int hidden_dim,
                              int batch,
                              hipStream_t stream)
{
    int total_count = batch * hidden_dim / 4;
    dim3 block_dims(1024);
    dim3 grid_dims((total_count - 1) / 1024 + 1);  // (batch_size);

    gptj_residual_add<<<grid_dims, block_dims, 0, stream>>>(
        input, output, attn, bias, total_count, hidden_dim / 4);
}

template void
launch_gptj_residual_add<float>(float*, float*, float*, float*, int, int, hipStream_t);
template void
launch_gptj_residual_add<__half>(__half*, __half*, __half*, __half*, int, int, hipStream_t);
